#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void task1() {
    int fac = 1;
    for (int i = 1; i <= 1 + threadIdx.x; i++) {
        fac *= i;
    }
    printf("%d!=%d\n", 1 + threadIdx.x, fac);
}

int main() {
    // initialize the number of threads and blocks
    const int numThreads = 8, numBlocks = 1;
    // run the function with no parameter
    task1<<<numBlocks, numThreads>>>();
    hipDeviceSynchronize();
    return 0;
}