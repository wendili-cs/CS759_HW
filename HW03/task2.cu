#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using std::cout, std::endl;

__global__ void task2(int *dA, int a) {
    dA[blockIdx.x * blockDim.x + threadIdx.x] = a * threadIdx.x + blockIdx.x;
}

int main() {
    // initialize the number of threads and blocks
    const int numThreads = 8, numBlocks = 2;
    int hA[numBlocks * numThreads], *dA;

    // allocate the memory
    hipMalloc((void **)&dA, sizeof(int) * numBlocks * numThreads);

    // generate a random integer
    srand((unsigned)time(NULL));
    const int RANGE = 100;
    int a = rand() % (RANGE + 1);

    // run function with 1 parameter
    task2<<<numBlocks, numThreads>>>(dA, a);
    hipDeviceSynchronize();

    // copy back to host from device
    hipMemcpy(hA, dA, sizeof(int) * numBlocks * numThreads,
               hipMemcpyDeviceToHost);

    // release memory in GPU
    hipFree(dA);

    for (int i = 0; i < numBlocks * numThreads; i++) {
        cout << hA[i] << " ";
    }
    cout << endl;
    return 0;
}