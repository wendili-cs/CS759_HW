#include "vscale.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using std::cout, std::endl;

float *a, *b, *da, *db;

int main(int argc, char **argv) {
    // initialize random functions
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> rand_a(-10.0, 10.0);
    std::uniform_real_distribution<float> rand_b(-1.0, 1.0);

    // initialize the array
    int n = std::atoi(argv[1]);
    a = (float *)malloc(n * sizeof(float));
    b = (float *)malloc(n * sizeof(float));
    for (int i = 0; i < n; i++) {
        a[i] = rand_a(random);
        b[i] = rand_b(random);
    }

    // allocate the memory
    hipMalloc((void **)&da, sizeof(float) * n);
    hipMalloc((void **)&db, sizeof(float) * n);

    // copy from host to device
    hipMemcpy(da, a, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * n, hipMemcpyHostToDevice);

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // run vscale function
    const int numThreads = 512, numBlocks = (n + numThreads - 1) / numThreads;
    vscale<<<numBlocks, numThreads>>>(da, db, n);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // copy back to host from device
    hipMemcpy(b, db, sizeof(float) * n, hipMemcpyDeviceToHost);

    // release memory in GPU
    hipFree(da);
    hipFree(db);

    cout << ms << endl;
    cout << b[0] << endl;
    cout << b[n - 1] << endl;
    return 0;
}