#include <cstddef>
#include <hip/hip_runtime.h>

using std::size_t;

__global__ void matmul_kernel(const float *A, const float *B, float *C,
                              size_t n) {
    size_t idx_C = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_C < n * n) {
        float tsum = 0.0;
        size_t idx_B = idx_C % n;
        size_t idx_A = idx_C - idx_B;
        for (size_t i = 0; i < n; i++) {
            tsum += A[idx_A + i] * B[idx_B + i * n];
        }
        C[idx_C] = tsum;
    }
};

void matmul(const float *A, const float *B, float *C, size_t n,
            unsigned int threads_per_block) {
    // run matmul kernel function
    const int numBlocks = (n * n + threads_per_block - 1) / threads_per_block;
    matmul_kernel<<<numBlocks, threads_per_block>>>(A, B, C, n);
    hipDeviceSynchronize();
};