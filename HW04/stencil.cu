#include <cstddef>
#include <hip/hip_runtime.h>
#include <stdio.h>

using std::size_t;

// // this is without shared memory version
// __global__ void stencil_kernel(const float *image, const float *mask,
//                                float *output, unsigned int n, unsigned int R)
//                                {
//     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < n) {
//         float o_i = 0.0;
//         for (unsigned int j = 0; j <= 2 * R; j++) {
//             int idx = i + j - R; // index of image
//             float img_v = (0 <= idx && idx < n) ? image[idx] : 1.0;
//             o_i += img_v * mask[j];
//         }
//         output[i] = o_i;
//     }
// };

__global__ void stencil_kernel(const float *image, const float *mask,
                               float *output, unsigned int n, unsigned int R) {
    // assign the varibles on shared memory
    extern __shared__ float sArray[];
    float *sMask = (float *)&sArray;
    float *spImage = (float *)&sMask[2 * R + 1];
    float *spOutput = (float *)&spImage[blockDim.x + 2 * R];

    // determine the index in global memory and shared memory
    int gIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int sIndex = threadIdx.x;

    // copy mask from global memory to shared memory
    if (sIndex < 2 * R + 1) {
        sMask[sIndex] = mask[sIndex];
    }
    // copy the image main part
    spImage[R + sIndex] = (gIndex < n) ? image[gIndex] : 1.0;

    // copy the padding part of the original image (each length R for two sides)
    if (sIndex < R) {
        // the corrsponding in-duty left index and right index on the original
        // image (to global image index)
        int imgLIndex = gIndex - R, imgRIndex = gIndex + blockDim.x;
        spImage[sIndex] = imgLIndex < 0 ? 1.0 : image[imgLIndex];
        spImage[R + blockDim.x + sIndex] =
            imgRIndex >= n ? 1.0 : image[imgRIndex];
    }
    // else if (R <= sIndex && sIndex < 2 * R) {
    //     // sIndex are from [R, 2R), imgIndex are from [x + 1, x + R],
    //     // where x is the last thread index of
    //     // this block {x + 1 = (blockIdx.x + 1) * blockDim.x}
    //     int imgIndex = (blockIdx.x + 1) * blockDim.x + sIndex - R;
    //     spImage[blockDim.x + R + sIndex] =
    //         (imgIndex) >= n ? 1.0 : image[imgIndex];
    // }
    // spOutput[sIndex] = 0.0;
    __syncthreads();

    // only when it is in scope
    if (gIndex < n) {
        float o_i = 0.0;
        for (size_t j = 0; j <= 2 * R; j++) {
            o_i += spImage[j + sIndex] * sMask[j];
        }
        spOutput[sIndex] = o_i;
        output[gIndex] = spOutput[sIndex];
    }
};

__host__ void stencil(const float *image, const float *mask, float *output,
                      unsigned int n, unsigned int R,
                      unsigned int threads_per_block) {
    // launch kernel function
    const int numBlocks = (n + threads_per_block - 1) / threads_per_block;
    stencil_kernel<<<numBlocks, threads_per_block,
                     (2 * threads_per_block + 4 * R + 1) * sizeof(float)>>>(
        image, mask, output, n, R);
    hipDeviceSynchronize();
};