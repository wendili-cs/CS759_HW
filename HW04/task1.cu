#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using std::cout, std::endl;

int main(int argc, char *argv[]) {
    // read the input of n and the number of threads per block
    int n = std::atoi(argv[1]), threads_per_block = std::atoi(argv[2]);

    // initialize random functions
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> rand_a(-1.0, 1.0);
    std::uniform_real_distribution<float> rand_b(-1.0, 1.0);

    float *A, *B, *C;

    // initialize the matrices A and B
    A = (float *)malloc(n * n * sizeof(float));
    B = (float *)malloc(n * n * sizeof(float));
    C = (float *)malloc(n * n * sizeof(float));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int idx = i * n + j;
            A[idx] = rand_a(random);
            B[idx] = rand_b(random);
        }
    }

    // allocate space in device
    float *dA, *dB, *dC;
    hipMalloc((void **)&dA, n * n * sizeof(float));
    hipMalloc((void **)&dB, n * n * sizeof(float));
    hipMalloc((void **)&dC, n * n * sizeof(float));

    // copy from host to device
    hipMemcpy(dA, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // run matmul function
    matmul(dA, dB, dC, n, threads_per_block);

    // Get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // copy back to the host
    hipMemcpy(C, dC, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // release memory in GPU
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    cout << C[n * n - 1] << endl;
    cout << ms << endl;

    return 0;
}