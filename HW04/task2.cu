#include "stencil.cuh"
#include <cstddef>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using std::cout, std::endl, std::size_t, std::atoi;

int main(int argc, char *argv[]) {
    // read input of n, R, threads
    size_t n = atoi(argv[1]), R = atoi(argv[2]),
           threads_per_block = atoi(argv[3]);

    // initialize random generator
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> rand_i(-1.0, 1.0);
    std::uniform_real_distribution<float> rand_m(-1.0, 1.0);

    // initialize the image, mask, and output
    float *image, *mask, *output;
    image = (float *)malloc(n * sizeof(float));
    mask = (float *)malloc((2 * R + 1) * sizeof(float));
    output = (float *)malloc(n * sizeof(float));

    // assign the image and mask
    for (size_t i = 0; i < n; i++) {
        image[i] = rand_i(random);
        // image[i] = 2.0;
    }
    for (size_t i = 0; i <= 2 * R; i++) {
        mask[i] = rand_m(random);
        // mask[i] = 0.5;
    }

    // allocate space in device
    float *dImage, *dMask, *dOutput;
    hipMalloc((void **)&dImage, n * sizeof(float));
    hipMalloc((void **)&dMask, (2 * R + 1) * sizeof(float));
    hipMalloc((void **)&dOutput, n * sizeof(float));

    // copy from host to device
    hipMemcpy(dImage, image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dMask, mask, (2 * R + 1) * sizeof(float),
               hipMemcpyHostToDevice);

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // run stencil function
    stencil(dImage, dMask, dOutput, n, R, threads_per_block);

    // Get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // copy back to host
    hipMemcpy(output, dOutput, n * sizeof(float), hipMemcpyDeviceToHost);

    // release memory in GPU
    hipFree(dImage);
    hipFree(dMask);
    hipFree(dOutput);

    // for DEBUG only
    // for (unsigned int i = 0; i < n; i++) {
    //     cout << output[i] << endl;
    // }
    cout << output[n - 1] << endl;
    cout << ms << endl;

    return 0;
}