#include <cstddef>
#include <hip/hip_runtime.h>
#include <iostream>

using std::size_t, std::cout, std::endl;

template <typename T>
__global__ void matmul_kernel(const T *A, const T *B, T *C, unsigned int n,
                              unsigned int block_dim) {
    // block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // index of the first/last element in the sub-matrix A
    int aBegin = n * block_dim * by;
    int aEnd = aBegin + n - 1;
    int aStep = block_dim;

    // index of the first/last element in the sub-matrix B
    int bBegin = block_dim * bx;
    int bStep = block_dim * n;

    // sub sum in C
    T Csub = (T)0;

    // allocate array on shared memory
    extern __shared__ int sdata[];
    T *As = (T *)sdata;
    T *Bs = (T *)&As[block_dim * block_dim];
    // __shared__ T As[block_dim][block_dim];
    // __shared__ T Bs[block_dim][block_dim];

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // load tiles from global
        int idxA = a + n * ty + tx, idxB = b + n * ty + tx;
        As[ty * block_dim + tx] = idxA < n * n ? A[idxA] : (T)0;
        Bs[ty * block_dim + tx] = idxB < n * n ? B[idxB] : (T)0;
        __syncthreads();

        for (int k = 0; k < block_dim; k++) {
            Csub += As[ty * block_dim + k] * Bs[k * block_dim + tx];
        }
        __syncthreads();
    }
    int c = n * block_dim * by + block_dim * bx;
    int idxC = c + n * ty + tx;
    if (idxC < n * n)
        C[idxC] = Csub;
}

__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n,
                       unsigned int block_dim) {
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x,
                 (n + dimBlock.y - 1) / dimBlock.y);
    unsigned int ssize = 2 * block_dim * block_dim * sizeof(int);
    matmul_kernel<int><<<dimGrid, dimBlock, ssize>>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}

__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n,
                       unsigned int block_dim) {
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x,
                 (n + dimBlock.y - 1) / dimBlock.y);
    unsigned int ssize = 2 * block_dim * block_dim * sizeof(float);
    matmul_kernel<float><<<dimGrid, dimBlock, ssize>>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}

__host__ void matmul_3(const double *A, const double *B, double *C,
                       unsigned int n, unsigned int block_dim) {
    dim3 dimBlock(block_dim, block_dim);
    dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x,
                 (n + dimBlock.y - 1) / dimBlock.y);
    unsigned int ssize = 2 * block_dim * block_dim * sizeof(double);
    matmul_kernel<double><<<dimGrid, dimBlock, ssize>>>(A, B, C, n, block_dim);
    hipDeviceSynchronize();
}