#include <cstddef>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using std::size_t, std::cout, std::endl;

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n) {
    // allocate array on shared memory
    extern __shared__ float sdata[];
    size_t tid = threadIdx.x;
    size_t i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    // first add two global values to one shared memory slot
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    __syncthreads();

    // apply reduction
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write the result back to global memory
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
};

__host__ void reduce(float **input, float **output, unsigned int N,
                     unsigned int threads_per_block) {
    // launch kernel function
    while (N > 1) {
        const int numBlocks =
            (N + 2 * threads_per_block - 1) / (2 * threads_per_block);
        reduce_kernel<<<numBlocks, threads_per_block,
                        threads_per_block * sizeof(float)>>>(
            (float *)input, (float *)output, N);
        hipDeviceSynchronize();
        // merge the results in blocks
        input = output;
        N = numBlocks;
    }
};