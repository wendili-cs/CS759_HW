#include "reduce.cuh"
#include <cassert>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using std::cout, std::endl, std::atoi;

int main(int argc, char *argv[]) {
    // read the array length N and the number of threads per block
    int N = atoi(argv[1]), threads_per_block = atoi(argv[2]);
    // assert the number of threads is 2**n
    assert((threads_per_block & (threads_per_block - 1)) == 0);

    // initialize random functions
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> rand(-1.0, 1.0);

    const int numBlocks =
        (N + 2 * threads_per_block - 1) / (2 * threads_per_block);
    float *input, *output;
    // hipHostMalloc(&input, N * sizeof(float));
    // hipHostMalloc(&output, numBlocks * sizeof(float));
    input = (float *)malloc(N * sizeof(float));
    output = (float *)malloc(numBlocks * sizeof(float));

    // initialize the array
    for (int i = 0; i < N; i++) {
        input[i] = rand(random);
        // input[i] = 1.0;
    }

    // allocate space in device
    float *dInput, *dOutput;
    hipMalloc((void **)&dInput, N * sizeof(float));
    hipMalloc((void **)&dOutput, numBlocks * sizeof(float));

    // copy memory from host to device
    hipMemcpy(dInput, input, N * sizeof(float), hipMemcpyHostToDevice);

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // run reduce function
    reduce((float **)dInput, (float **)dOutput, N, threads_per_block);

    // Get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // copy back the result to the host
    hipMemcpy(output, dOutput, numBlocks * sizeof(float),
               hipMemcpyDeviceToHost);

    // DEBUG ONLY
    // for(int i = 0; i < numBlocks; i++)
    //     cout << "output[" << i << "] = " << output[i] << endl;

    // release memory in GPU
    hipFree(dInput);
    hipFree(dOutput);

    cout << output[0] << endl;
    cout << ms << endl;

    return 0;
}