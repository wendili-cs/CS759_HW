#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <type_traits>

using std::cout, std::endl, std::atoi;

template <typename T> void init_matrix(T *M, unsigned int n) {
    // initialize random functions
    // std::default_random_engine random(time(NULL));
    // std::uniform_real_distribution<T> rand(-1.0, 1.0);

    int L = 3; // distribuction between [-L, L]
    // initialize the matrix
    for (unsigned int i = 0; i < n; i++) {
        for (unsigned int j = 0; j < n; j++) {
            unsigned int idx = i * n + j;
            // M[idx] = rand(random);
            M[idx] = (T)(rand() % (2 * L + 1) - L);
        }
    }
}

template <typename T>
void carry_test(unsigned int n, unsigned int block_dim,
                void (*mm)(const T *, const T *, T *, unsigned int,
                           unsigned int)) {
    // allocate space in host
    T *A = (T *)malloc(n * n * sizeof(T));
    T *B = (T *)malloc(n * n * sizeof(T));
    T *C = (T *)malloc(n * n * sizeof(T));

    // initialize the matrix
    init_matrix<T>(A, n);
    init_matrix<T>(B, n);
    init_matrix<T>(C, n);

    // allocate space in device
    T *dA, *dB, *dC;
    hipMalloc((void **)&dA, n * n * sizeof(T));
    hipMalloc((void **)&dB, n * n * sizeof(T));
    hipMalloc((void **)&dC, n * n * sizeof(T));

    // copy memory from host to device
    hipMemcpy(dA, A, n * n * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n * n * sizeof(T), hipMemcpyHostToDevice);

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // run matmul function
    mm(dA, dB, dC, n, block_dim);

    // Get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // copy back the result to the host
    hipMemcpy(C, dC, n * n * sizeof(T), hipMemcpyDeviceToHost);

    // release memory in GPU
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    cout << C[0] << endl;
    cout << C[n * n - 1] << endl;
    cout << ms << endl;
}

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]), block_dim = atoi(argv[2]);

    carry_test<int>(n, block_dim, matmul_1);
    carry_test<float>(n, block_dim, matmul_2);
    carry_test<double>(n, block_dim, matmul_3);

    return 0;
}