#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

void mmul(hipblasHandle_t handle, const float *A, const float *B, float *C,
          int n) {
    const float a = 1, b = 2;
    const float *alpha = &a, *beta = &b;
    // do the general matrix multiplication
    hipblasStatus_t stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
                                      alpha, A, n, B, n, beta, C, n);
    hipDeviceSynchronize();
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("Warning: mmul function failed!");
    }
};