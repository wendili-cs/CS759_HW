#include <cmath>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void HillisSteele_scan_kernel(float *g_odata, const float *g_idata,
                                         unsigned int n, int shift) {
    // allocate shared memroy
    extern volatile __shared__ float temp[];

    int thid = threadIdx.x;
    int pout = 0, pin = 1;
    if (thid >= n)
        temp[thid] = 0;
    else {
        float first_ele = (shift == 0) ? 0 : g_odata[shift - 1];
        temp[thid] = (thid == 0) ? first_ele : g_idata[shift + thid - 1];
    }
    __syncthreads();

    for (int offset = 1; offset < n; offset <<= 1) {
        pout = 1 - pout;
        pin = 1 - pout;

        if (thid >= offset) {
            temp[pout * n + thid] =
                temp[pin * n + thid] + temp[pin * n + thid - offset];
        } else {
            temp[pout * n + thid] = temp[pin * n + thid];
        }
        __syncthreads();
    }
    if (thid < n)
        g_odata[shift + thid] = g_idata[shift + thid] + temp[pout * n + thid];
}

__host__ void scan(const float *input, float *output, unsigned int n,
                   unsigned int threads_per_block) {
    const int numBlocks = (n + threads_per_block - 1) / threads_per_block;
    for (int i = 0; i < numBlocks; i++) {
        int shift = i * threads_per_block;
        int len =
            (threads_per_block > n - shift) ? n - shift : threads_per_block;
        // printf("len:%d\n", len);
        HillisSteele_scan_kernel<<<1, threads_per_block,
                                   2 * threads_per_block * sizeof(float)>>>(
            output, input, len, shift);
        hipDeviceSynchronize();
    }
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        // printf(cudaGetErrorString(error));
    }
}
