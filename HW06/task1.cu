#include "mmul.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using std::cout, std::endl, std::atoi;

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]), n_tests = atoi(argv[2]);

    // allocate memory for matrices
    float *A, *B, *C;
    hipMallocManaged(&A, n * n * sizeof(float));
    hipMallocManaged(&B, n * n * sizeof(float));
    hipMallocManaged(&C, n * n * sizeof(float));

    // initialize random functions
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> rand(-1.0, 1.0);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int idx = i * n + j;
            A[idx] = rand(random);
            B[idx] = rand(random);
            C[idx] = rand(random);
            // A[idx] = 1.0;
            // B[idx] = 1.0;
            // C[idx] = 1.0;
        }
    }

    // create handle
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // repeatly run mmul function
    for (int i = 0; i < n_tests; i++) {
        mmul(handle, A, B, C, n);
    }

    // get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // destory the handler
    hipblasDestroy(handle);

    cout << ms / n_tests << endl;

    return 0;
}