#include "scan.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

using std::cout, std::endl, std::atoi;

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]), threads_per_block = atoi(argv[2]);
    float *input, *output;
    hipMallocManaged(&input, n * sizeof(float));
    hipMallocManaged(&output, n * sizeof(float));

    // initialize random functions
    std::default_random_engine random(time(NULL));
    std::uniform_real_distribution<float> rand(-1.0, 1.0);

    for (int i = 0; i < n; i++) {
        input[i] = rand(random);
        // input[i] = 1;
    }

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // run kernel function
    scan(input, output, n, threads_per_block);

    // get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    cout << output[n - 1] << endl;
    cout << ms << endl;

    return 0;
}