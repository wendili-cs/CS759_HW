#define HIPCUB_STDERR // print CUDA runtime errors to console
#include <cub/device/device_reduce.cuh>
#include <cub/util_allocator.cuh>
#include <iostream>
#include <random>
#include <stdio.h>
using namespace hipcub;
CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

using std::cout, std::endl, std::atoi;

int main(int argc, char *argv[]) {
    int num_items = atoi(argv[1]);
    // Set up host arrays
    float *h_in = (float *)malloc(num_items * sizeof(float));
    ;

    // initialize random functions
    std::default_random_engine eng(time(NULL));
    std::uniform_real_distribution<float> rand(-1.0, 1.0);

    // initialize the array
    for (int i = 0; i < num_items; i++) {
        h_in[i] = rand(eng);
        // h_in[i] = 1.0;
    }

    // Set up device arrays
    float *d_in = NULL;
    g_allocator.DeviceAllocate((void **)&d_in, sizeof(float) * num_items);

    // Initialize device input
    hipMemcpy(d_in, h_in, sizeof(float) * num_items, hipMemcpyHostToDevice);

    // Setup device output array
    float *d_sum = NULL;
    g_allocator.DeviceAllocate((void **)&d_sum, sizeof(float) * 1);

    // Request and allocate temporary storage
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum,
                      num_items);
    g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Do the actual reduce operation
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum,
                      num_items);

    // Get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // obtain the sum result
    float gpu_sum;
    hipMemcpy(&gpu_sum, d_sum, sizeof(float) * 1, hipMemcpyDeviceToHost);

    cout << gpu_sum << endl;
    cout << ms << endl;

    // Cleanup
    if (d_in)
        g_allocator.DeviceFree(d_in);
    if (d_sum)
        g_allocator.DeviceFree(d_sum);
    if (d_temp_storage)
        g_allocator.DeviceFree(d_temp_storage);

    return 0;
}