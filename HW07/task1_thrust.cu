#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>

using std::cout, std::endl, std::atoi;

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]);
    thrust::host_vector<float> arr(n);

    // initialize random functions
    thrust::default_random_engine eng(1234);
    thrust::uniform_real_distribution<float> rand(-1.0, 1.0);

    // assign random floats
    thrust::generate(arr.begin(), arr.end(), [&] { return rand(eng); });

    // transfer data to device
    thrust::device_vector<float> d_arr = arr;

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // apply reduce function
    float res =
        thrust::reduce(d_arr.begin(), d_arr.end(), 0.0, thrust::plus<float>());

    // Get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    cout << res << endl;
    cout << ms << endl;

    return 0;
}