#include "count.cuh"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>

using std::cout, std::endl, std::atoi;

int main(int argc, char *argv[]) {
    int n = atoi(argv[1]);
    thrust::host_vector<int> in(n);
    thrust::device_vector<int> values;
    thrust::device_vector<int> counts;

    // initialize random functions
    thrust::default_random_engine eng(time(NULL));
    thrust::uniform_int_distribution<int> rand(0, 500);

    // assign random floats
    thrust::generate(in.begin(), in.end(), [&] { return rand(eng); });

    // transfer data to device
    thrust::device_vector<int> d_in = in;

    // ready for timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // run the count function
    count(d_in, values, counts);

    // Get the elapsed time in milliseconds
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    cout << values.back() << endl;
    cout << counts.back() << endl;
    cout << ms << endl;

    return 0;
}